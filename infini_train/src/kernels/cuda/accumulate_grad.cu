#include "hip/hip_runtime.h"
#include "infini_train/include/dispatcher.h"
#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

__global__ void AccumulateGradKernel(const float *grad_ptr, float rate, float *tensor_ptr, size_t num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        tensor_ptr[idx] += rate * grad_ptr[idx];
    }
}

void AccumulateGrad(const std::shared_ptr<Tensor> &gradient, float rate, const std::shared_ptr<Tensor> &tensor) {
    size_t num_elements = gradient->NumElements();

    const float *grad_ptr = static_cast<const float *>(gradient->DataPtr());
    float *tensor_ptr = static_cast<float *>(tensor->DataPtr());

    int threads_per_block = 256;
    int num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;

    AccumulateGradKernel<<<num_blocks, threads_per_block>>>(grad_ptr, rate, tensor_ptr, num_elements);
}

__global__ void AdamAccumulateGradKernel(const float *grad_ptr, float *param, float *m, float *v, float learning_rate,
                    float beta1,float beta2, float beta1_pow, float beta2_pow, float eps, size_t num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        float L = grad_ptr[idx];
        m[idx] = beta1 * m[idx] + (1 - beta1) * L;
        v[idx] = beta2 * v[idx] + (1 - beta2) * L * L;
        float m_hat = m[idx] / (1 - beta1_pow);
        float v_hat = v[idx] / (1 - beta2_pow);
        param[idx] -= learning_rate * m_hat / sqrtf(v_hat + eps);
    }
}

void AdamAccumulateGrad(const std::shared_ptr<Tensor> &grad, const std::shared_ptr<Tensor> &param,
                        const std::shared_ptr<Tensor> &m, const std::shared_ptr<Tensor> &v, float learning_rate,
                        float beta1, float beta2, float eps, int64_t t) {
    // =================================== 作业 ===================================
    // TODO：实现Adam优化器的梯度累积和参数更新
    // REF:
    // =================================== 作业 ===================================
    size_t num_elements = grad->NumElements();
    float beta1_pow = std::pow(beta1, t);
    float beta2_pow = std::pow(beta2, t);

    const float *grad_ptr = static_cast<const float *>(grad->DataPtr());
    float *param_ptr = static_cast<float *>(param->DataPtr());
    float *m_ptr = static_cast<float *>(m->DataPtr());
    float *v_ptr = static_cast<float *>(v->DataPtr());

    int threads_per_block = 256;
    int num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;

    AdamAccumulateGradKernel<<<num_blocks, threads_per_block>>>(grad_ptr, param_ptr, m_ptr, v_ptr, learning_rate,
         beta1, beta2, beta1_pow, beta2_pow, eps, num_elements);
}
} // namespace infini_train::kernels::cuda

#define REGISTER_CUDA_ACCUMULATE_GRAD_KERNEL(kernel_name)                                                              \
    REGISTER_KERNEL(infini_train::DeviceType::kCUDA, kernel_name, infini_train::kernels::cuda::kernel_name)

REGISTER_CUDA_ACCUMULATE_GRAD_KERNEL(AccumulateGrad)
REGISTER_CUDA_ACCUMULATE_GRAD_KERNEL(AdamAccumulateGrad)

#undef REGISTER_CUDA_ACCUMULATE_GRAD_KERNEL
