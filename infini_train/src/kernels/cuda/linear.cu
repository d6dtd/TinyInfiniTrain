#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "glog/logging.h"
#include <cub/block/block_reduce.cuh>

#include "infini_train/include/dispatcher.h"
#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t status = call;                                                                                     \
        if (status != hipSuccess) {                                                                                   \
            LOG(FATAL) << "CUDA Error: " << hipGetErrorString(status) << " at " << __FILE__ << ":" << __LINE__;       \
        }                                                                                                              \
    } while (0)

#define CUBLAS_CHECK(call)                                                                                             \
    do {                                                                                                               \
        hipblasStatus_t status = call;                                                                                  \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                                                         \
            LOG(FATAL) << "CUBLAS Error: " << cublasGetStatusString(status) << " at " << __FILE__ << ":" << __LINE__;  \
        }                                                                                                              \
    } while (0)

std::shared_ptr<Tensor> MatmulForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &other) {
    // =================================== 作业 ===================================
    // TODO：实现CUDA上的矩阵乘法前向计算
    // REF:
    // =================================== 作业 ===================================

    const auto &input_dims = input->Dims();
    const auto &other_dims = other->Dims();
    CHECK_GE(input_dims.size(), 2);
    CHECK_GE(other_dims.size(), 2);

    const int64_t M1 = input_dims[input_dims.size() - 2];
    const int64_t N1 = input_dims[input_dims.size() - 1];
    const int64_t M2 = other_dims[other_dims.size() - 2];
    const int64_t N2 = other_dims[other_dims.size() - 1];

    // M1 x N1 * M2 x N2 --> M1 x N2
    CHECK(N1 == M2);

    const int64_t size = std::accumulate(input_dims.rbegin() + 2, input_dims.rend(), 1, std::multiplies<int64_t>{});

    auto output_dims = input_dims;
    output_dims[output_dims.size() - 1] = N2;
    auto output = std::make_shared<Tensor>(output_dims, DataType::kFLOAT32, input->GetDevice());
    output->Fill<float>(0.0f);

    const float alpha = 1.0f;
    const float beta = 0.0f;
    std::vector<hipStream_t> streams(size);
    std::vector<hipblasHandle_t> handles(size);

    for (int i = 0; i < size; ++i) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
        CUBLAS_CHECK(hipblasCreate(&handles[i]));
        CUBLAS_CHECK(hipblasSetStream(handles[i], streams[i]));

        // output = input * other --> output.T = other.T * input.T
        // C = output.T[N2, M1]
        // A = other.T[N2, M2]
        // B = input.T[N1, M1]
        CUBLAS_CHECK(hipblasSgemm(handles[i], HIPBLAS_OP_N, HIPBLAS_OP_N, N2, M1, N1, &alpha,
            static_cast<const float *>(other->DataPtr()) + i * N2 * M2, N2,
            static_cast<const float *>(input->DataPtr()) + i * N1 * M1, N1,
            &beta, static_cast<float *>(output->DataPtr()) + i * N2 * M1, N2));
    }
    // 等待所有 stream 完成
    hipDeviceSynchronize();

    // 清理资源
    for (int i = 0; i < size; ++i) {
        CUBLAS_CHECK(hipblasDestroy(handles[i]));
        hipStreamDestroy(streams[i]);
    }
    return output;
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
MatmulBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &other,
               const std::shared_ptr<Tensor> &grad_output) {
    // =================================== 作业 ===================================
    // TODO：实现CUDA上的矩阵乘法反向传播
    // REF:
    // =================================== 作业 ===================================

    const auto &input_dims = input->Dims();
    const auto &other_dims = other->Dims();
    CHECK_GE(input_dims.size(), 2);
    CHECK_GE(other_dims.size(), 2);

    auto grad_input = std::make_shared<Tensor>(input_dims, DataType::kFLOAT32, input->GetDevice());
    auto grad_other = std::make_shared<Tensor>(other_dims, DataType::kFLOAT32, other->GetDevice());
    grad_input->Fill<float>(0.0f);
    grad_other->Fill<float>(0.0f);

    const int64_t M = input_dims[input_dims.size() - 2];
    const int64_t K = input_dims[input_dims.size() - 1];
    const int64_t N = other_dims[other_dims.size() - 1];

    // M x K * K x N --> M x N
    CHECK(K == other_dims[other_dims.size() - 2]);

    const int64_t size = std::accumulate(input_dims.rbegin() + 2, input_dims.rend(), 1, std::multiplies<int64_t>{});

    const float alpha = 1.0f;
    const float beta = 0.0f;
    std::vector<hipStream_t> streams(size);
    std::vector<hipblasHandle_t> handles(size);

    for (int i = 0; i < size; ++i) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
        CUBLAS_CHECK(hipblasCreate(&handles[i]));
        CUBLAS_CHECK(hipblasSetStream(handles[i], streams[i]));

        // grad_input = grad_output * other.T
        // grad_input.T = other * grad_output.T
        // C = grad_input.T[K, M]
        // A = other.T[N, K]
        // B = grad_output.T[N, M]

        CUBLAS_CHECK(hipblasSgemm(handles[i], HIPBLAS_OP_T, HIPBLAS_OP_N, K, M, N, &alpha,
            static_cast<const float *>(other->DataPtr()) + i * N * K, N,
            static_cast<const float *>(grad_output->DataPtr()) + i * N * M, N,
            &beta, static_cast<float *>(grad_input->DataPtr()) + i * K * M, K));

        // grad_other = input.T * grad_output
        // grad_other.T = grad_output.T * input
        // C = grad_other.T[N, K]
        // A = grad_output.T[N, M]
        // B = input.T[K, M]
       CUBLAS_CHECK(hipblasSgemm(handles[i], HIPBLAS_OP_N, HIPBLAS_OP_T, N, K, M, &alpha,
            static_cast<const float *>(grad_output->DataPtr()) + i * N * M, N,
            static_cast<const float *>(input->DataPtr()) + i * K * M, K,
            &beta, static_cast<float *>(grad_other->DataPtr()) + i * N * K, N));
    }
    // 等待所有 stream 完成
    hipDeviceSynchronize();

    // 清理资源
    for (int i = 0; i < size; ++i) {
        CUBLAS_CHECK(hipblasDestroy(handles[i]));
        hipStreamDestroy(streams[i]);
    }

    return {grad_input, grad_other};
}

__global__ void BiasCopyKernel(float *output, const float *bias, int bs, int out_features) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= bs * out_features) {
        return;
    }
    int j = idx % out_features;
    output[idx] = bias[j];
}

std::shared_ptr<Tensor> LinearForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight,
                                      bool transpose, const std::shared_ptr<Tensor> &bias) {

    /*
        !transpose: output = input * weight + bias
        output[*, out_features] = input[*, in_features] * weight[in_features, out_features] + bias[out_features]

        transpose:  output = input * weight^T + bias
        output[*, out_features] = input[*, in_features] * weight[out_features, in_features]^T + bias[out_features]
    */

    const auto &input_dims = input->Dims();
    CHECK_GE(input_dims.size(), 2);
    const int64_t bs = std::accumulate(input_dims.rbegin() + 1, input_dims.rend(), 1, std::multiplies<int64_t>{});
    const int64_t in_features = *input_dims.rbegin();

    const auto &weight_dims = weight->Dims();
    CHECK_EQ(weight_dims.size(), 2);
    CHECK_EQ(in_features, weight_dims[transpose ? 1 : 0]);

    // As for cublas:
    // C = alpha * op(B) * op(A) + beta * C
    // Dimensions:
    //   input:  (bs, in_features)
    //   weight: (in_features, out_features) or (out_features, in_features) if transposed
    //   output: (bs, out_features)
    const int64_t out_features = weight_dims[transpose ? 0 : 1];

    auto output_dims = input_dims;
    *output_dims.rbegin() = out_features;
    auto output = std::make_shared<Tensor>(output_dims, DataType::kFLOAT32, input->GetDevice());

    if (bias) {
        CHECK_EQ(bias->Dims().size(), 1);
        CHECK_EQ(bias->Dims()[0], out_features);
        int threads_per_block = 256;
        int num_blocks = (bs * out_features + threads_per_block - 1) / threads_per_block;
        BiasCopyKernel<<<num_blocks, threads_per_block>>>(
            static_cast<float *>(output->DataPtr()), static_cast<const float *>(bias->DataPtr()), bs, out_features);
    } else {
        output->Fill<float>(0.0f);
    }

    const float alpha = 1.0f;
    const float beta = 1.0f;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    if (transpose) {
        // weight is [out_features, in_features] here

        // output = input * weight.T --> output.T = weight * input.T
        // C = output.T[out_features, bs]
        // A = weight.T[in_features, out_features]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, out_features, bs, in_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), in_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(output->DataPtr()), out_features));
    } else {
        // output = input * weight --> output.T =  weight.T * input.T
        // C = output.T[out_features, bs]
        // A = weight.T[out_features, in_features]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, out_features, bs, in_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), out_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(output->DataPtr()), out_features));
    }
    CUBLAS_CHECK(hipblasDestroy(handle));
    return output;
}

template <int BLOCK_SIZE>
__global__ void ReduceColumnsKernel(const float *__restrict__ input, float *__restrict__ output, int num_rows,
                                    int num_cols) {
    using BlockReduce = hipcub::BlockReduce<float, BLOCK_SIZE>;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int row = blockIdx.x;
    float sum = 0.0f;

    for (int col = threadIdx.x; col < num_cols; col += blockDim.x) { sum += input[row * num_cols + col]; }

    float reduced = BlockReduce(temp_storage).Sum(sum);

    if (threadIdx.x == 0) {
        output[row] = reduced;
    }
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
LinearBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight, bool transpose,
               int64_t out_features, const std::shared_ptr<Tensor> &grad_output, const bool bias) {
    const auto &input_dims = input->Dims();
    CHECK_GE(input_dims.size(), 2);
    const int64_t bs = std::accumulate(input_dims.rbegin() + 1, input_dims.rend(), 1, std::multiplies<int64_t>{});
    const int64_t in_features = *input_dims.rbegin();

    const auto &weight_dims = weight->Dims();
    CHECK_EQ(weight_dims.size(), 2);
    CHECK_EQ(in_features, weight_dims[transpose ? 1 : 0]);
    CHECK_EQ(out_features, weight_dims[transpose ? 0 : 1]);

    auto grad_input = std::make_shared<Tensor>(input_dims, DataType::kFLOAT32, grad_output->GetDevice());
    auto grad_weight = std::make_shared<Tensor>(weight_dims, DataType::kFLOAT32, grad_output->GetDevice());
    grad_input->Fill<float>(0.0f);
    grad_weight->Fill<float>(0.0f);
    std::shared_ptr<Tensor> grad_bias = nullptr;
    if (bias) {
        grad_bias = std::make_shared<Tensor>(std::vector<int64_t>{out_features}, DataType::kFLOAT32,
                                             grad_output->GetDevice());
        grad_bias->Fill<float>(0.0f);
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    if (transpose) {
        // weight is [out_features, in_features] here

        // d_input = d_output * weight --> d_input.T = weight.T * d_output.T
        // C = d_input.T[in_features, bs]
        // A = weight.T[in_features, out_features]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, in_features, bs, out_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), in_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_input->DataPtr()), in_features));

        // d_weight = d_output.T * input --> d_weight.T = input.T * d_output
        // C = d_weight.T[in_features, out_features]
        // A = input.T[in_features, bs]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, in_features, out_features, bs, &alpha,
                                 static_cast<const float *>(input->DataPtr()), in_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_weight->DataPtr()), in_features));
    } else {
        // weight is [in_features, out_features] here

        // d_input = d_output * weight.T --> d_input.T = weight * d_output.T
        // C = d_input.T[in_features, bs]
        // A = weight.T[out_features, in_features]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, in_features, bs, out_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), out_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_input->DataPtr()), in_features));

        // d_weight = input.T * d_output --> d_weight.T = d_output.T * input
        // C = d_weight.T[out_features, in_features]
        // A = d_output.T[out_features, bs]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, out_features, in_features, bs, &alpha,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(grad_weight->DataPtr()), out_features));
    }

    // d_bias = \sum_i(i=0, bs-1) d_output[i]
    if (bias) {
        constexpr int BLOCK_SIZE = 256;
        int threads_per_block = BLOCK_SIZE;
        int num_blocks = out_features;
        ReduceColumnsKernel<BLOCK_SIZE>
            <<<num_blocks, threads_per_block>>>(static_cast<const float *>(grad_output->DataPtr()),
                                                static_cast<float *>(grad_bias->DataPtr()), out_features, bs);
    }

    CUBLAS_CHECK(hipblasDestroy(handle));

    return {grad_input, grad_weight, grad_bias};
}
} // namespace infini_train::kernels::cuda

#define REGISTER_CUDA_LINEAR_KERNEL(kernel_name)                                                                       \
    REGISTER_KERNEL(infini_train::DeviceType::kCUDA, kernel_name, infini_train::kernels::cuda::kernel_name)

REGISTER_CUDA_LINEAR_KERNEL(MatmulForward)
REGISTER_CUDA_LINEAR_KERNEL(MatmulBackward)
REGISTER_CUDA_LINEAR_KERNEL(LinearForward)
REGISTER_CUDA_LINEAR_KERNEL(LinearBackward)

#undef REGISTER_CUDA_LINEAR_KERNEL
